#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>
#include <iostream>

__device__ __host__
float_t* sub3d(float_t v1[3], float_t v2[3], float_t result[3]) {

    for (int r = 0; r < 3; r++){
        result[r] = v1[r] - v2[r];
    }
    return result;
}

__device__ __host__
float_t* add3d(float_t v1[3], float_t v2[3], float_t result[3]) {

    for (int r = 0; r < 3; r++){
        result[r] = v1[r] + v2[r];
    }
    return result;
}

__device__ __host__
float_t* scaler_mult3d(float_t vector[3], float_t scalar, float_t result[3]) {
    for (int r = 0; r < 3; r++){
        result[r] = vector[r] * scalar;
    }
    return result;
}

__device__ __host__
float_t dot3d(float_t v1[3], float_t v2[3]) {
    float_t result = 0;
    for (int r = 0; r < 3; r++){
        float_t prod = v1[r] * v2[r];
        result = result + prod;
    }    
    return result;
}

__device__
float_t* cross3d(float_t v1[3], float_t v2[3], float_t result[3]) {
    result[0] = v1[1] * v2[2] - v1[2] * v2[1];
    result[1] = v1[2] * v2[0] - v1[0] * v2[2];
    result[3] = v1[0] * v2[1] - v1[1] * v2[0];
    return result;
}


// So we have a tensor full of coordinates corresponding to all the vertices in the mesh ---> [V, 3 (x, y, z)]
// for each face we will also have indices into the vertices tensor. -----------------------> [F, 3 (v1, v2, v3)]
__global__
void find_intersection_kernel(
    int n_rays, int n_faces, 
    at::PackedTensorAccessor32<float_t, 2> vertex_acc,
    at::PackedTensorAccessor32<int32_t, 2> face_acc, 
    at::PackedTensorAccessor32<float_t, 2> origin_acc,
    at::PackedTensorAccessor32<float_t, 2> direct_acc,
    at::PackedTensorAccessor32<float_t, 2> results) {

    int ray_ix = blockIdx.x * blockDim.x + threadIdx.x;
    int face_ix = blockIdx.y * blockDim.y + threadIdx.y;

    //note: all pointers are meant to represent 3d in the below code
    if (ray_ix < n_rays) {
        if (face_ix < n_faces) {

            // printf("(%d, %d)\n", ray_ix, face_ix);
        
            // vectors: ray origin and direction
            float_t *ray_direction_ptr = &direct_acc[ray_ix][0];
            float_t *ray_origin_ptr = &origin_acc[ray_ix][0];

            // vectors: vertices of current face
            float_t *v0_ptr = &vertex_acc[face_acc[face_ix][0]][0];
            float_t *v1_ptr = &vertex_acc[face_acc[face_ix][1]][0];
            float_t *v2_ptr = &vertex_acc[face_acc[face_ix][2]][0];
            
            // get edge vectors of current face
            float_t edge1[3];
            float_t edge2[3];
            sub3d(v1_ptr, v0_ptr, edge1);
            sub3d(v2_ptr, v0_ptr, edge2);

            // determinant of matrix A for eqn Ax = b
            float_t h[3];
            cross3d(ray_direction_ptr, edge2, h);

            // printf("(%d, %d) - value of e1 vector: [%f, %f, %f]\n", ray_ix, face_ix, edge1[0], edge1[1], edge1[2]);
            // printf("(%d, %d) - value of h vector: [%f, %f, %f]\n", ray_ix, face_ix,  h[0], h[1], h[2]);

            float_t determinant = dot3d(edge1, h);
            float_t inv_det = 1 / determinant;

            // printf("Value of determinant: %f\n", determinant);

            // // no solution if determinant is zero. 
            // if (determinant > -1e4 && determinant < 1e4) { 
            //     results[ray_ix][face_ix] = std::numeric_limits<float_t>::infinity();
            //     return;
            // }

            // // solve for first barycentric coordinate, u
            // float_t s[3];
            // sub3d(ray_origin_ptr, v0_ptr, s);
            // float_t u = inv_det * dot3d(s, h);

            // // solve for second barycentric coordinate, v
            // float_t q[3];
            // cross3d(s, edge1, q);
            // float_t v = inv_det * dot3d(ray_direction_ptr, q);

            // // validate barycentric coordinates
            // if (v < 0.0 || u + v > 1) {
            //     printf("invalid coordinates: %d, %d \n", u, v);
            //     results[ray_ix][face_ix] = std::numeric_limits<float_t>::infinity();
            //     return;
            // }

            // // calculate distance from ray origin to intersection (t)
            // float t = inv_det * dot3d(edge2, q);
            results[ray_ix][face_ix] = 0;
            // printf("Exiting index: (%d, %d)\n", ray_ix, face_ix);
        }
    }
}

__host__
at::Tensor measure_distance_cuda(
    at::Tensor vertices,
    at::Tensor faces,
    at::Tensor ray_origins,
    at::Tensor ray_directions
) {

    int n_rays = ray_directions.size(0);
    int n_faces = faces.size(0);

    at::Tensor distances = at::zeros({n_rays, n_faces});

    dim3 blocks(16, 16);
    dim3 threads(32, 32);

    find_intersection_kernel<<<blocks, threads>>>(
        n_rays, n_faces,
        vertices.packed_accessor32<float_t, 2>(),
        faces.packed_accessor32<int32_t, 2>(),
        ray_origins.packed_accessor32<float_t, 2>(),
        ray_directions.packed_accessor32<float_t, 2>(),
        distances.packed_accessor32<float_t, 2>()
    );

    hipDeviceSynchronize();
    return distances;
}

